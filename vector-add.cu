#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>

// CPU function
void addVectors(const float *a, const float *b, float *c, int n){
    for(int i = 0; i < n; i++){
        c[i] = a[i] + b[i];
    }
    return;
}

int main(){
    float *a, *b, *c;
    int n = 10;
    size_t size = n*sizeof(float);
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    for(int i=0; i<n; i++){
        a[i] = i;
        b[i] = i;
    }

    addVectors(a, b, c, n);

    std::cout << std::endl;
    for(int i=0; i<n; i++)
        std::cout<< c[i] << " ";

    free(a);
    free(b);
    free(c);

    return 0;
}