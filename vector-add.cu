#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

// CPU function
void addVectors(const float *a, const float *b, float *c, int n)
{
    for (int i = 0; i < n; i++)
    {
        c[i] = a[i] + b[i];
    }
    return;
}

int main()
{
    float *a, *b, *c;
    int n = 1e9; // n = 1 billion
    size_t size = n * sizeof(float);
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);

    auto start = std::chrono::high_resolution_clock::now(); // Start the timer

    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    addVectors(a, b, c, n);
    auto end = std::chrono::high_resolution_clock::now(); // Stop the timer

    std::chrono::duration<double> diff = end - start;

    std::cout << n / 1e9 << " billion elements took : " << diff.count() << " seconds" << std::endl;
    std::cout << "Result of 10 numbers in c[] : " << std::endl;
    for (int i = 0; i < 10; i++)
        std::cout << c[i] << " ";
    std::cout << std::endl;

    free(a);
    free(b);
    free(c);

    return 0;
}