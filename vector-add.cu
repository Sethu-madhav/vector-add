#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>

// CPU function
void addVectors(float *a, float *b, float *c, int n){
    for(int i = 0; i < n; i++){
        c[i] = a[i] + b[i];
    }
    return;
}

int main(){
    float *a, *b, *c;
    int n = 10;
    size_t size = n*sizeof(float);
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    for(int i=0; i<n; i++){
        a[i] = i;
        b[i] = i;
    }

    addVectors(a, b, c, n);

    printf("\n");
    for(int i=0; i<n; i++)
        std::cout<< c[i] << " ";

    return 0;
}